#include "hip/hip_runtime.h"

#include "kernel.h"

using namespace std;
using namespace cv;
using namespace cv::gpu;

struct result{
	int fern[10];
	int res;
	float conf;
};

struct proData{
	int size, nstructs, structSize, featureSizeSize;
	float var, thr_fern;
};

struct boundingbox{
	int x;
	int y;
	int width; 
	int height;
	int sidx;
};
struct feature{
	uchar x1, y1, x2, y2;
};

__global__ void kernel(float* posteriors, feature* featureData, proData * dt, boundingbox* bbdata, const PtrStepSz<uchar1> img, const PtrStepSz<float1> iisum, PtrStep<float1> iisqsum, result *result)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < (*dt).size){
		result[i].res = 0;
		int fern[10];
		feature featureTmp;
		float brs = iisum(bbdata[i].y + bbdata[i].height, bbdata[i].x + bbdata[i].width).x;
		float bls = iisum(bbdata[i].y + bbdata[i].height, bbdata[i].x).x;
		float trs = iisum(bbdata[i].y, bbdata[i].x + bbdata[i].width).x;
		float tls = iisum(bbdata[i].y, bbdata[i].x).x;
		float brsq = iisqsum(bbdata[i].y + bbdata[i].height, bbdata[i].x + bbdata[i].width).x;
		float blsq = iisqsum(bbdata[i].y + bbdata[i].height, bbdata[i].x).x;
		float trsq = iisqsum(bbdata[i].y, bbdata[i].x + bbdata[i].width).x;
		float tlsq = iisqsum(bbdata[i].y, bbdata[i].x).x;
		float mean = (brs + tls - trs - bls) / ((float)bbdata[i].height*(float)bbdata[i].width);
		float sqmean = (brsq + tlsq - trsq - blsq) / ((float)bbdata[i].height*(float)bbdata[i].width);
		if (sqmean - mean*mean >= (*dt).var){
			int leaf;
			for (int t = 0; t<(*dt).nstructs; t++){
				leaf = 0;
				for (int f = 0; f<(*dt).structSize; f++){
					featureTmp = featureData[bbdata[i].sidx*(*dt).featureSizeSize + t*(*dt).structSize + f];
					leaf = (leaf << 1) + (img(bbdata[i].y + featureTmp.y1, bbdata[i].x + featureTmp.x1).x > img(bbdata[i].y+featureTmp.y2, bbdata[i].x+featureTmp.x2).x);
				}
				fern[t] = leaf;
			}
			float votes = 0;
			for (int t = 0; t <(*dt).nstructs; t++) {
				votes += posteriors[t*(int)pow(2.0f, (float)(*dt).structSize)+ fern[t]];
			}
			result[i].conf = votes;
			for (int t = 0; t<10; t++){
				result[i].fern[t] = fern[t];
			}
			if (votes>(*dt).nstructs*(*dt).thr_fern){
				result[i].res = 1;
			}
		}
		else{
			result[i].conf = 0.0;
		}
	}
}

void safeCall(hipError_t e){
	if (e != hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(e));
		system("pause");
	}
}

void processWithCuda(TLD *tld, const Mat &img, const FerNNClassifier &classifier){
	GpuMat gpuiisum, gpuiisqsum, gpuimg;
	gpuiisum.upload(tld->iisum);
	gpuiisum.convertTo(gpuiisum, CV_32F);
	gpuiisqsum.upload(tld->iisqsum);
	gpuiisqsum.convertTo(gpuiisqsum, CV_32F);
	gpuimg.upload(img);
	vector<BoundingBox> grid = tld->grid;
	proData dt;
	dt.size = grid.size();
	dt.var = tld->var;
	dt.nstructs = classifier.nstructs;
	dt.structSize = classifier.structSize;
	dt.thr_fern = classifier.thr_fern;
	boundingbox * data = new boundingbox[dt.size];
	for (int i = 0; i < dt.size; i++){
		data[i].x = grid[i].x;
		data[i].y = grid[i].y;
		data[i].width = grid[i].width;
		data[i].height = grid[i].height;
		data[i].sidx = grid[i].sidx;
	}
	int featureSize = classifier.features.size();
	int featureSizeSize = classifier.features[0].size();
	dt.featureSizeSize = featureSizeSize;
	feature * featureData = new feature[featureSize*featureSizeSize];
	for (int i = 0; i < featureSize; i++){
		for (int j = 0; j < featureSizeSize; j++){
			featureData[i*featureSizeSize + j].x1 = classifier.features[i][j].x1;
			featureData[i*featureSizeSize + j].y1 = classifier.features[i][j].y1;
			featureData[i*featureSizeSize + j].x2 = classifier.features[i][j].x2;
			featureData[i*featureSizeSize + j].y2 = classifier.features[i][j].y2;
		}
	}
	boundingbox *dev_bbdata = NULL;
	feature *dev_featureData = NULL;
	proData *dev_dt = NULL;
	float * dev_posteriors = NULL;
	int posteriorsSize = classifier.posteriors.size();
	int posteriorsSizeSize = classifier.posteriors[0].size();
	float *posteriors = new float[posteriorsSize*posteriorsSizeSize];
	for (int i = 0; i < posteriorsSize; i++){
		for (int j = 0; j < posteriorsSizeSize; j++){
			posteriors[i*posteriorsSizeSize + j] = classifier.posteriors[i][j];
		}
	}
	safeCall(hipMalloc((void**)&dev_posteriors, posteriorsSize*posteriorsSizeSize*sizeof(float)));
	safeCall(hipMalloc((void**)&dev_featureData, featureSize*featureSizeSize*sizeof(feature)));
	safeCall(hipMalloc((void**)&dev_bbdata, dt.size*sizeof(boundingbox)));
	safeCall(hipMalloc((void**)&dev_dt, sizeof(proData)));
	safeCall(hipMemcpy(dev_posteriors, posteriors, posteriorsSize*posteriorsSizeSize*sizeof(float), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_dt, &dt, sizeof(proData), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_bbdata, data, dt.size*sizeof(boundingbox), hipMemcpyHostToDevice));
	safeCall(hipMemcpy(dev_featureData, featureData, featureSize*featureSizeSize*sizeof(feature), hipMemcpyHostToDevice));

	result *dev_result = NULL;
	result *res = new result[dt.size];
	safeCall(hipMalloc((void**)&dev_result, dt.size*sizeof(result)));
	kernel << <1024, 1024 >> >(dev_posteriors, dev_featureData, dev_dt, dev_bbdata, gpuimg, gpuiisum, gpuiisqsum, dev_result);
	safeCall(hipMemcpy(res, dev_result, dt.size * sizeof(result), hipMemcpyDeviceToHost));

	int output = 0;
	for (int i = 0; i < dt.size; i++){
		if (res[i].res)
			output++;
	}
	printf("%d %d\n", dt.size, output);

	for (int i = 0; i < dt.size; i++){
		tld->tmp.conf[i] = res[i].conf;
		for (int j = 0; j < 10; j++){
			tld->tmp.patt[i][j] = res[i].fern[j];
		}
		if (res[i].res){
			tld->dt.bb.push_back(i);
		}
	}

	safeCall(hipFree(dev_posteriors));
	safeCall(hipFree(dev_bbdata));
	safeCall(hipFree(dev_dt));
	safeCall(hipFree(dev_featureData));
	safeCall(hipFree(dev_result));
	
	delete data;
	delete featureData;
	delete res;

	return;
}